#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    int c;
    int *dev_c;
    err = hipMalloc( (void**) &dev_c, sizeof(int) );
    
    if (err != hipSuccess) {
        printf("Memorry alloc error!");
    }

    add <<<1,1>>>( 2, 7, dev_c );

    err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        printf("cuda Memcpy error!");
    }

    printf("%d", c);

    hipFree(dev_c);

    return 0;
}
