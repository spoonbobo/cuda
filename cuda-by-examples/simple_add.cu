#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    int c;
    int *dev_c;
    cout << "Pointer value (before allocating memory on GPU): " << dev_c << endl;
    cout << "Pointer address: " <<  &dev_c << endl;

    err = hipMalloc( (void**) &dev_c, sizeof(int) );

    cout << "Pointer value (after allocating memory on GPU): " << dev_c << endl;
    cout << "Pointer address: " << &dev_c << endl;
    
    if (err != hipSuccess) {
        printf("Memorry alloc error!");
    }

    add <<<1,1>>>( 2, 7, dev_c );

    err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        printf("cuda Memcpy error!");
    }

    printf("%d", c);

    hipFree(dev_c);

    return 0;
}
