#include <stdio.h>
#include <hip/hip_runtime.h>

#define N (33 * 1024)
#define N_THREADS 64

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}

void cpu_vector_add(int *a, int *b, int *c) {
    int tid = 0;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid++;
    }
}

__global__ void gpu_vector_add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void) {
    int a[N], b[N], c[N], c_gpu[N];
    int *dev_a, *dev_b, *dev_c;

    // define threads
    dim3 threads(N_THREADS);
    dim3 blocks((N+N_THREADS-1)/N_THREADS);

    // allocate memory
    handleError(hipMalloc( (void**)&dev_a, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_b, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_c, N * sizeof(int) ));
    
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i*i;
    }

    // copy host to device
    handleError(hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ));
    handleError(hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ));

    gpu_vector_add<<<blocks, threads>>>(dev_a, dev_b, dev_c);

    // copy device to host
    handleError(hipMemcpy( c_gpu, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ));

    cpu_vector_add(a, b, c);

    printf("CPU simple vector add version: \n");

    for (int i=N-10; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    printf("GPU simple vector add version: \n");

    for (int i=N-10; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c_gpu[i]);
    }

    return 0;
}