#include <stdio.h>
#include <hip/hip_runtime.h>

#define N (33 * 1024)
#define threadsPerBlock 64

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}

void cpu_vector_add(int *a, int *b, int *c) {
    int tid = 0;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid++;
    }
}

__global__ void gpu_vector_add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void) {
    int *a, *b, *c, *c_gpu;
    int *dev_a, *dev_b, *dev_c;

    // define threads
    dim3 threads(threadsPerBlock);
    dim3 blocks((N+threadsPerBlock-1)/threadsPerBlock);

    // allocate memory to cpu
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    c_gpu = (int *)malloc(N * sizeof(int));

    // allocate memory to gpu
    handleError(hipMalloc( (void**)&dev_a, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_b, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_c, N * sizeof(int) ));
    
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i*i;
    }

    // copy host to device
    handleError(hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ));
    handleError(hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ));

    gpu_vector_add<<<blocks, threads>>>(dev_a, dev_b, dev_c);

    // copy device to host
    handleError(hipMemcpy( c_gpu, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ));

    cpu_vector_add(a, b, c);

    printf("CPU simple vector add version: \n");

    for (int i=N-10; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    printf("GPU simple vector add version: \n");

    for (int i=N-10; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c_gpu[i]);
    }

    // free GPU mem
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    // free CPU mem
    free(a);
    free(b);
    free(c);
    free(c_gpu);

    return 0;
}