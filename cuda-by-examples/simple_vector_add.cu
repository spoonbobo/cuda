#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}

void cpu_vector_add(int *a, int *b, int *c) {
    int tid = 0;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid++;
    }
}

__global__ void gpu_vector_add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[N], b[N], c[N], c_gpu[N];
    int *dev_a, *dev_b, *dev_c;

    // define threads
    dim3 threads(N);

    // allocate memory
    handleError(hipMalloc( (void**)&dev_a, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_b, N * sizeof(int) ));
    handleError(hipMalloc( (void**)&dev_c, N * sizeof(int) ));
    
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i*i;
    }

    // copy host to device
    handleError(hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ));
    handleError(hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ));

    gpu_vector_add<<<1, threads>>>(dev_a, dev_b, dev_c);

    // copy device to host
    handleError(hipMemcpy( c_gpu, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ));

    cpu_vector_add(a, b, c);

    printf("CPU simple vector add version: \n");

    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    printf("GPU simple vector add version: \n");

    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c_gpu[i]);
    }

    return 0;
}