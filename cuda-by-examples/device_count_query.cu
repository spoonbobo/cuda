#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    hipError_t err = hipSuccess;
    int count;
    err = hipGetDeviceCount (& count) ;
    printf("%d", count);
    return 0;
}
