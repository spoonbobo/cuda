#include <stdio.h>
#include <hip/hip_runtime.h>

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}