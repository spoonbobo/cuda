#include <stdio.h>
#include <hip/hip_runtime.h>

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}

int main(void) {
    hipError_t err = hipSuccess;
    int count;
    handleError( hipGetDeviceCount (& count) );

    hipDeviceProp_t prop;
    for (int i=0; i<count; i++) {
        handleError( hipGetDeviceProperties(&prop, i) );
        printf("   --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device copy overlap: ");
        if (prop.deviceOverlap) {
            printf("Enabled\n");
        } else {
            printf("Disabled\n");
        }
        printf("Kernel execution timeout: ");
        if (prop.kernelExecTimeoutEnabled) {
            printf("Enabled\n");
        } else {
            printf("Disabled\n");
        }

        printf("   --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %ld\n", prop.totalGlobalMem);
        printf("Total constant mem: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment: %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions, (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    return 0;
}