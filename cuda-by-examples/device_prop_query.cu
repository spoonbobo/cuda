#include <stdio.h>
#include <hip/hip_runtime.h>

void handleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s", hipGetErrorString(err));
        exit(err);
    }
}

int main(void) {
    hipError_t err = hipSuccess;
    int count;
    handleError( hipGetDeviceCount (& count) );

    hipDeviceProp_t prop;
    for (int i=0; i<count; i++) {
        handleError( hipGetDeviceProperties(&prop, i) );
        printf("   --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    }
    return 0;
}